
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define SIZE 50000
 void printArr( int arr[], int n )
{
    int i;
    for ( i = 0; i < n; ++i )
        printf( "%d ", arr[i] );
}
__device__ int d_size;

__global__ void partition (int *arr, int *arr_l, int *arr_h, int n)
{
    int z = blockIdx.x*blockDim.x+threadIdx.x;
    d_size = 0;
    __syncthreads();
    if (z<n)
      {
        int h = arr_h[z];
        int l = arr_l[z];
        int x = arr[h];
        int i = (l - 1);
        int temp;
        for (int j = l; j <= h- 1; j++)
          {
            if (arr[j] <= x)
              {
                i++;
                temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
              }
          }
        temp = arr[i+1];
        arr[i+1] = arr[h];
        arr[h] = temp;
        int p = (i + 1);
        if (p-1 > l)
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = l;
            arr_h[ind] = p-1;  
          }
        if ( p+1 < h )
          {
            int ind = atomicAdd(&d_size, 1);
            arr_l[ind] = p+1;
            arr_h[ind] = h; 
          }
      }
}
 
void quickSortIterative (int arr[], int l, int h)
{
    int lstack[ h - l + 1 ], hstack[ h - l + 1];
 
    int top = -1, *d_d, *d_l, *d_h;
 
    lstack[ ++top ] = l;
    hstack[ top ] = h;

    hipMalloc(&d_d, (h-l+1)*sizeof(int));
    hipMemcpy(d_d, arr,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&d_l, (h-l+1)*sizeof(int));
    hipMemcpy(d_l, lstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc(&d_h, (h-l+1)*sizeof(int));
    hipMemcpy(d_h, hstack,(h-l+1)*sizeof(int),hipMemcpyHostToDevice);
    int n_t = 1;
    int n_b = 1;
    int n_i = 1; 
    while ( n_i > 0 )
    {
        partition<<<n_b,n_t>>>( d_d, d_l, d_h, n_i);
        int answer;
        hipMemcpyFromSymbol(&answer, HIP_SYMBOL(d_size), sizeof(int), 0, hipMemcpyDeviceToHost); 
        if (answer < 1024)
          {
            n_t = answer;
          }
        else
          {
            n_t = 1024;
            n_b = answer/n_t + (answer%n_t==0?0:1);
          }
        n_i = answer;
        hipMemcpy(arr, d_d,(h-l+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
}
 

 
int main(int argc, char **argv) {

      
       int *arr;
       int numElements; 
    
    FILE *inp1 = fopen(argv[1], "r");
    fscanf(inp1, "%d", &numElements);
    
    printf("\nInput length = %d\n",numElements);
    arr= new int[numElements];
    for(int i = 0; i < numElements; ++i){
	fscanf(inp1, "%d", &arr[i]);
	
    }
    
   /* printf("\nInput\n");    
    for(int i=0;i<numElements;i++){
    	printf("%d ",arr[i]);
    }
*/
    int start_s=clock();
    quickSortIterative( arr, 0, numElements);
    int stop_s=clock();
    
    
  
    
    
    
    FILE *op = fopen(argv[2], "r");
    fscanf(op, "%d", &numElements);
    int *output;
    output=new int[numElements];
    for(int i = 0; i < numElements; ++i){
	fscanf(op, "%d", &output[i]);
    }
    int flag=0;
    printf("\n");
    for(int i=0;i<numElements;i++){
    	if(output[i]!=arr[i+1]){
    		printf("\nSolution wrong Expecting : %d but got : %d\n",output[i],arr[i]);
    		flag=1;
    	}
    }
    if(flag==0){
    printf("\nSolution is Correct !!!\n");
    printf("\nTime :  %f s \n",(stop_s-start_s)/double(CLOCKS_PER_SEC));
    }
    
    fclose(op);
    fclose(inp1);
    
    //printf("\nOutput\n");
    //printArr( arr, numElements);

    return 0;
}
